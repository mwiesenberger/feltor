#include "hip/hip_runtime.h"
#include <iostream>

#include "dg/algorithm.h"
#include "ds.h"

double sine(double x, double y, double z){return sin(z);}
double cosine(double x, double y, double z){return cos(z);}

double func(double x, double y, double z)
{
    double r2 = x*x+y*y;
    return r2*sin(z);
}
double deri(double x, double y, double z)
{
    double r2 = x*x+y*y;
    return r2*cos(z);
}
double r2( double x, double y) {return x*x+y*y;}
double r2z( double x, double y, double z) {return (x*x+y*y)*z;}


int main()
{
    std::cout << "# Test straight field lines and boundaries in z.\n";
    std::cout << "# Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    std::cout <<"# You typed\n"
              <<"n:  "<<n<<"\n"
              <<"Nx: "<<Nx<<"\n"
              <<"Ny: "<<Ny<<"\n"
              <<"Nz: "<<Nz<<std::endl;
    dg::CartesianGrid3d g3d( -1, 1, -1, 1, 0.1, M_PI+0.1, n, Nx, Ny, Nz, dg::DIR, dg::DIR, dg::NEU);
    dg::CartesianGrid2d perp_grid( -1, 1, -1, 1, n, Nx, Ny, dg::DIR, dg::DIR);
    const dg::DVec w3d = dg::create::volume( g3d);
    dg::Timer t;
    t.tic();
    dg::geo::CylindricalVectorLvl1 vec( dg::geo::Constant(0),
            dg::geo::Constant(0), dg::geo::Constant(1), dg::geo::Constant(1),
            dg::geo::Constant(1));

    dg::geo::DS<dg::CartesianGrid3d, dg::IDMatrix, dg::DVec> ds (
            vec, g3d, dg::DIR, dg::DIR, dg::geo::FullLimiter());
    t.toc();
    std::cout << "# Creation of parallel Derivative took     "<<t.diff()<<"s\n";

    dg::DVec function = dg::evaluate( func, g3d), derivative(function);
    dg::DVec constfunc = dg::evaluate( sine, g3d);
    const dg::DVec solution = dg::evaluate( deri, g3d);
    const dg::DVec constsolution = dg::evaluate( cosine, g3d);
    t.tic();
    ds.set_boundaries( dg::DIR, sin(g3d.z0()),sin(g3d.z1()));
    ds.ds(dg::centered, constfunc, derivative);
    t.toc();
    std::cout << "Straight:\n";
    std::cout << "# Application of parallel Derivative took  "<<t.diff()<<"s\n";
    dg::blas1::axpby( 1., constsolution, -1., derivative);
    double norm = dg::blas2::dot( constsolution, w3d, constsolution);
    double diff = sqrt( dg::blas2::dot( derivative, w3d, derivative)/norm );
    std::cout << "    DIR const:   "<< diff<<"\n";
    t.tic();
    ds.set_boundaries( dg::NEU, cos(g3d.z0()),cos(g3d.z1()));
    ds.ds(  dg::centered, constfunc, derivative);
    t.toc();
    std::cout << "# Application of parallel Derivative took  "<<t.diff()<<"s\n";
    dg::blas1::axpby( 1., constsolution, -1., derivative);
    diff = sqrt( dg::blas2::dot( derivative, w3d, derivative)/norm );
    std::cout << "    NEU const:   "<< diff << "\n";

    t.tic();
    dg::DVec left = dg::evaluate( r2, perp_grid), right(left);
    dg::blas1::scal( left, sin(g3d.z0()));
    dg::blas1::scal( right, sin(g3d.z1()));
    ds.set_boundaries( dg::DIR, left,right);
    ds.ds( dg::centered, function, derivative);
    t.toc();
    std::cout << "# Application of parallel Derivative took  "<<t.diff()<<"s\n";
    dg::blas1::axpby( 1., solution, -1., derivative);
    diff = sqrt( dg::blas2::dot( derivative, w3d, derivative)/norm );
    std::cout << "    DIR l/r:     "<< diff << "\n";
    t.tic();
    dg::DVec global = dg::evaluate( r2z, g3d);
    ds.set_boundaries( dg::DIR, global, sin(g3d.z0())/(g3d.z0()+g3d.hz()/2.), sin(g3d.z1())/(g3d.z1()-g3d.hz()/2.));
    ds.ds( dg::centered, function, derivative);
    t.toc();
    std::cout << "# Application of parallel Derivative took  "<<t.diff()<<"s\n";
    dg::blas1::axpby( 1., solution, -1., derivative);
    diff = sqrt( dg::blas2::dot( derivative, w3d, derivative)/norm );
    std::cout << "    DIR global:  "<< diff <<"\n";

    return 0;
}
