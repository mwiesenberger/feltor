#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <mpi.h>
#include "dg/algorithm.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "ds.h"
#include "toroidal.h"

const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius

int main(int argc, char* argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz, mx[2], max_iter = 1e4;
    std::string method = "dg";
    unsigned letters = 0;
    MPI_Comm comm;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    if(rank==0)std::cout << "# Test the parallel derivative DS in cylindrical coordinates for circular flux surfaces with DIR and NEU boundary conditions.\n";
    dg::mpi_init3d( dg::NEU, dg::NEU, dg::PER, n, Nx, Ny, Nz, comm);
    if( rank == 0)
    {
        std::cout <<"# You typed\n"
                  <<"n:  "<<n<<"\n"
                  <<"Nx: "<<Nx<<"\n"
                  <<"Ny: "<<Ny<<"\n"
                  <<"Nz: "<<Nz<<std::endl;
        std::cout <<"# Type mx (10) and my (10)\n";
        std::cin >> mx[0] >> mx[1];
        std::cout << "# You typed\n"
                  <<"mx: "<<mx[0]<<"\n"
                  <<"my: "<<mx[1]<<std::endl;
        std::cout << "# Type method (dg, nearest, linear, cubic) \n";
        std::cin >> method;
        method.erase( std::remove( method.begin(), method.end(), '"'), method.end());
        letters = method.size();
        std::cout << "# You typed\n"
                  <<"method: "<< method<<std::endl;
        std::cout << "# Create parallel Derivative!\n";
    }
    MPI_Bcast( mx, 2, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast( &letters, 1, MPI_INT, 0, MPI_COMM_WORLD);
    method.resize( letters);
    MPI_Bcast( &method[0], letters, MPI_CHAR, 0, MPI_COMM_WORLD);

    const dg::CylindricalMPIGrid3d g3d( R_0-a, R_0+a, -a, a, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::NEU, dg::NEU, dg::PER, comm);
    //create magnetic field
    const dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
    auto bhat = dg::geo::createBHat(mag);
    //create Fieldaligned object and construct DS from it
    dg::geo::Fieldaligned<dg::aProductMPIGeometry3d,dg::MIDMatrix,dg::MDVec>
        dsFA( bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx[0],
                mx[1], -1, method);
    dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIDMatrix,
        dg::MDVec> ds( dsFA);
    ///##########################################################///
    auto ff = dg::geo::TestFunctionDirNeu(mag);
    dg::MDVec fun = dg::evaluate( ff, g3d);
    dg::MDVec derivative(fun);
    dg::MDVec divb = dg::evaluate( dg::geo::Divb(mag), g3d);
    dg::MDVec sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::TestFunctionDirNeu>(mag,ff), g3d);
    dg::MDVec sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::TestFunctionDirNeu>(mag,ff), g3d);
    dg::MDVec sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::TestFunctionDirNeu>(mag,ff), g3d);
    dg::MDVec sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::TestFunctionDirNeu>(mag,ff), g3d);
    dg::MDVec sol4 =dg::evaluate( dg::geo::OMDsDivDsFunction<dg::geo::TestFunctionDirNeu>(mag,ff), g3d);
    std::vector<std::pair<std::string, std::array<const dg::MDVec*,2>>> names{
         {"forward",{&fun,&sol0}},          {"backward",{&fun,&sol0}},
         {"forward2",{&fun,&sol0}},         {"backward2",{&fun,&sol0}},
         {"centered",{&fun,&sol0}},         {"centered_bc_along",{&fun,&sol0}},
         {"dss",{&fun,&sol1}},              {"dss_bc_along",{&fun,&sol1}},
         {"divForward",{&fun,&sol2}},       {"divBackward",{&fun,&sol2}},
         {"divCentered",{&fun,&sol2}},      {"directLap",{&fun,&sol3}},
         {"directLap_bc_along",{&fun,&sol3}}, {"invCenteredLap",{&sol4,&fun}}
    };
    if(rank==0)std::cout << "# TEST NEU Boundary conditions!\n";
    if(rank==0)std::cout << "# TEST ADJOINT derivatives do unfortunately not fulfill Neumann BC!\n";
    ///##########################################################///
    if(rank==0)std::cout <<"Neumann:\n";
    dg::MDVec vol3d = dg::create::volume( g3d);
    for( const auto& tuple :  names)
    {
        std::string name = std::get<0>(tuple);
        const dg::MDVec& function = *std::get<1>(tuple)[0];
        const dg::MDVec& solution = *std::get<1>(tuple)[1];
        callDS( ds, name, function, derivative, max_iter,1e-8);
        double sol = dg::blas2::dot( vol3d, solution);
        dg::blas1::axpby( 1., solution, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    if(rank==0)std::cout << "# Reconstruct parallel derivative!\n";
    dsFA.construct( bhat, g3d, dg::DIR, dg::DIR, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1], -1, method);
    ds.construct( dsFA);
    if(rank==0)std::cout << "# TEST DIR Boundary conditions!\n";
    ///##########################################################///
    if(rank==0)std::cout << "Dirichlet: \n";
    for( const auto& tuple :  names)
    {
        std::string name = std::get<0>(tuple);
        const dg::MDVec& function = *std::get<1>(tuple)[0];
        const dg::MDVec& solution = *std::get<1>(tuple)[1];
        callDS( ds, name, function, derivative, max_iter,1e-8);
        double sol = dg::blas2::dot( vol3d, solution);
        dg::blas1::axpby( 1., solution, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }

    ///##########################################################///
    if(rank==0)std::cout << "TEST FIELDALIGNED EVALUATION of a Gaussian\n";
    dg::Gaussian init0(R_0+0.5, 0, 0.2, 0.2, 1);
    dg::GaussianZ modulate(0., M_PI/3., 1);
    dg::Timer t;
    t.tic();
    dg::MDVec aligned = dg::geo::fieldaligned_evaluate( g3d, bhat, init0,
            modulate, Nz/2, 2);
    t.toc();
    if(rank==0)std::cout << "# took "<<t.diff()<<"s\n";
    ds.ds( dg::centered, aligned, derivative);
    double norm = dg::blas2::dot(vol3d, derivative);
    if(rank==0)std::cout << "# Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_t)\n";
    t.tic();
    aligned = dsFA.evaluate( init0, modulate, Nz/2, 2);
    t.toc();
    if(rank==0)std::cout << "# took "<<t.diff()<<"s\n";
    ds.ds( dg::centered, aligned, derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    if(rank==0)std::cout << "# Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_mpit)\n";
    ///##########################################################///
    if(rank==0)std::cout << "# TEST STAGGERED GRID DERIVATIVE\n";
    dg::MDVec zMinus(fun), eMinus(fun), zPlus(fun), ePlus(fun);
    dg::MDVec funST(fun);
    dg::geo::Fieldaligned<dg::aProductMPIGeometry3d,dg::MIDMatrix,dg::MDVec>  dsFAST(
            bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1],
            g3d.hz()/2., method);
    if(rank==0)std::cout <<"Time: ";
    t.tic();
    for( unsigned i=0; i<10; i++)
        ds.centered( fun, derivative);
    t.toc();
    double gbytes=fun.size()*sizeof(double)/1e9;
    if(rank==0)std::cout << t.diff()/10 << " #\t "<<gbytes*83*10/t.diff()<<"GB/s\n" ;
    for( auto bc : {dg::NEU, dg::DIR})
    {
        if( bc == dg::DIR)
            if(rank==0)std::cout << "DirichletST:\n";
        if( bc == dg::NEU)
            if(rank==0)std::cout << "NeumannST:\n";
        dsFAST( dg::geo::zeroMinus, fun, zMinus);
        dsFAST( dg::geo::einsPlus,  fun, ePlus);
        dg::geo::assign_bc_along_field_1st( dsFAST, zMinus, ePlus, zMinus, ePlus,
            bc, {0,0});
        dg::geo::ds_average( dsFAST, 1., zMinus, ePlus, 0., funST);
        dsFAST( dg::geo::zeroPlus, funST, zPlus);
        dsFAST( dg::geo::einsMinus, funST, eMinus);
        dg::geo::assign_bc_along_field_1st( dsFAST, eMinus, zPlus, eMinus, zPlus,
            bc, {0,0});
        dg::geo::ds_slope( dsFAST, 1., eMinus, zPlus, 0., derivative);
        double sol = dg::blas2::dot( vol3d, sol0);
        dg::blas1::axpby( 1., sol0, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        std::string name = "forward";
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";

        // now try the adjoint direction (should be exactly the same result)
        dsFAST( dg::geo::zeroPlus, fun, zPlus);
        dsFAST( dg::geo::einsMinus, fun, eMinus);
        dg::geo::assign_bc_along_field_1st( dsFAST, eMinus, zPlus, eMinus, zPlus,
            bc, {0,0});
        dg::geo::ds_average( dsFAST, 1., eMinus, zPlus, 0., funST);
        dsFAST( dg::geo::einsPlus, funST, ePlus);
        dsFAST( dg::geo::zeroMinus, funST, zMinus);
        dg::geo::assign_bc_along_field_1st( dsFAST, zMinus, ePlus, zMinus, ePlus,
            bc, {0,0});
        dg::geo::ds_slope( dsFAST, 1., zMinus, ePlus, 0., derivative);
        dg::blas1::axpby( 1., sol0, -1., derivative);
        norm = dg::blas2::dot( derivative, vol3d, derivative);
        name = "backward";
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    MPI_Finalize();
    return 0;
}
