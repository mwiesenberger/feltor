#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "mpi.h"
#include "dg/algorithm.h"
#include "testfunctors.h"
#include "ds.h"
#include "guenter.h"
#include "flux.h"
#include "toroidal.h"
#include "mpi_curvilinear.h"


int main(int argc, char * argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz, mx[2];
    MPI_Comm comm;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    if(rank==0)std::cout << "# Test DS on flux grid (No Boundary conditions)!\n";
    dg::mpi_init3d( dg::DIR, dg::PER, dg::PER, n, Nx, Ny, Nz, comm);
    if( rank == 0)
    {
        std::cout <<"# You typed\n"
                  <<"n:  "<<n<<"\n"
                  <<"Nx: "<<Nx<<"\n"
                  <<"Ny: "<<Ny<<"\n"
                  <<"Nz: "<<Nz<<std::endl;
        std::cout <<"# Type mx (1) and my (100)\n";
        std::cin >> mx[0] >> mx[1];
        std::cout << "# You typed\n"
                  <<"mx: "<<mx[0]<<"\n"
                  <<"my: "<<mx[1]<<std::endl;
        std::cout << "# Create parallel Derivative!\n";
    }
    MPI_Bcast( mx, 2, MPI_INT, 0, MPI_COMM_WORLD);
    const double R_0 = 3;
    const double I_0 = 10; //q factor at r=1 is I_0/R_0
    const double a  = 1; //small radius
    const dg::geo::TokamakMagneticField mag = dg::geo::createGuenterField(R_0, I_0);
    double psi_0 = -20, psi_1 = -4;
    dg::Timer t;
    t.tic();
    dg::geo::FluxGenerator flux( mag.get_psip(), mag.get_ipol(), psi_0, psi_1, R_0, 0., 1);
    if(rank==0)std::cout << "# Constructing Grid..."<<std::endl;
    dg::geo::CurvilinearProductMPIGrid3d g3d(flux, n, Nx, Ny,Nz, dg::NEU, dg::PER, dg::PER, comm);
    if(rank==0)std::cout << "# Constructing Fieldlines..."<<std::endl;
    dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIDMatrix,
        dg::MDVec> ds( mag, g3d, dg::NEU, dg::PER, dg::geo::FullLimiter(),
                1e-8, mx[0], mx[1]);

    t.toc();
    if(rank==0)std::cout << "# Construction took "<<t.diff()<<"s\n";
    ///##########################################################///
    //(MIND THE PULLBACK!)
    auto ff = dg::geo::TestFunctionPsi2(mag,a);
    const dg::MDVec fun = dg::pullback( ff, g3d);
    dg::MDVec derivative(fun);
    dg::MDVec sol0 = dg::pullback( dg::geo::DsFunction<dg::geo::TestFunctionPsi2>(mag,ff), g3d);
    dg::MDVec sol1 = dg::pullback( dg::geo::DssFunction<dg::geo::TestFunctionPsi2>(mag,ff), g3d);
    dg::MDVec sol2 = dg::pullback( dg::geo::DsDivFunction<dg::geo::TestFunctionPsi2>(mag,ff), g3d);
    dg::MDVec sol3 = dg::pullback( dg::geo::DsDivDsFunction<dg::geo::TestFunctionPsi2>(mag,ff), g3d);
    dg::MDVec sol4 = dg::pullback( dg::geo::OMDsDivDsFunction<dg::geo::TestFunctionPsi2>(mag,ff), g3d);
    std::vector<std::pair<std::string, std::array<const dg::MDVec*,2>>> names{
         {"forward",{&fun,&sol0}},          {"backward",{&fun,&sol0}},
         {"centered",{&fun,&sol0}},         {"dss",{&fun,&sol1}},
         {"divForward",{&fun,&sol2}},       {"divBackward",{&fun,&sol2}},
         {"divCentered",{&fun,&sol2}},      {"directLap",{&fun,&sol3}},
         {"invCenteredLap",{&sol4,&fun}}
    };
    ///##########################################################///
    if(rank==0)std::cout <<"Flux:\n";
    const dg::MDVec vol3d = dg::create::volume( g3d);
    for( const auto& tuple :  names)
    {
        std::string name = std::get<0>(tuple);
        const dg::MDVec& function = *std::get<1>(tuple)[0];
        const dg::MDVec& solution = *std::get<1>(tuple)[1];
        callDS( ds, name, function, derivative, g3d.size(),1e-8);
        double sol = dg::blas2::dot( vol3d, solution);
        dg::blas1::axpby( 1., solution, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        double vol = dg::blas1::dot( vol3d, derivative)/sqrt( dg::blas2::dot( vol3d, function)); // using function in denominator makes entries comparable
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n"
                  <<"    "<<name+"_vol:"<<std::setw(30-name.size())
                  <<" "<<vol<<"\n";
    }
    ///##########################################################///
    MPI_Finalize();
    return 0;
}
