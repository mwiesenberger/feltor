#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "dg/algorithm.h"


#include "lanczos.h"
#include "mcg.h"
#include "matrixfunction.h"
#include "matrixsqrt.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::DIR;
// const double m=3./2.;
// const double n=4.;
// const double m=1./2.;
// const double n=1.;
const double m=1./2.;
const double n=1.;
const double ms=1./2.;
const double ns=2.;
const double alpha = 1./2.;
const double ell_fac = (m*m+n*n);
const double ell_facs = (ms*ms+ns*ns);

const double amp=10.0;
const double bgamp=1.0;

double lhs( double x, double y){ return sin(x*m)*sin(y*n);}
double lhss( double x, double y){ return sin(x*ms)*sin(y*ns);}
double sin2( double x, double y){ return amp*sin(x*m)*sin(y*n)*sin(x*m)*sin(y*n);}
using Matrix = dg::DMatrix;
using Container = dg::DVec;

int main(int argc, char * argv[])
{
    dg::Timer t;

    unsigned n, Nx, Ny;
    std::cout << "# Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout <<"# You typed\n"
              <<"n:  "<<n<<"\n"
              <<"Nx: "<<Nx<<"\n"
              <<"Ny: "<<Ny<<std::endl;
    unsigned iter = 0;

    unsigned max_iter = 1;
    std::cout << "# Type max_iter of tridiagonalization (500) ?\n";
    std::cin >> max_iter ;
    std::cout << "# Type in eps of tridiagonalization (1e-7)\n";
    double eps = 1e-7; //# of pcg iter increases very much if
    std::cin >> eps;
    std::cout <<"# You typed\n"
              <<"max_iter: "<<max_iter<<"\n"
              <<"eps: "<<eps<<std::endl;

    double erel = 0;

    dg::Grid2d g( 0, lx, 0, ly,n, Nx, Ny, bcx, bcy);
    const Container w2d = dg::create::weights( g);
    Container w2d_scaled(w2d);

    double max_weights = dg::blas1::reduce(w2d, 0., dg::AbsMax<double>() );
    double min_weights = dg::blas1::reduce(w2d, max_weights, dg::AbsMin<double>() );
    std::cout << "#   min(W)  = "<<min_weights <<"  max(W) = "<<max_weights << "\n";
    dg::Elliptic<dg::CartesianGrid2d, Matrix, Container> A( {g, dg::centered, 1.0});

    std::vector<std::string> outs = {
            "K_0(-alpha A)",
            "K_0(d, -alpha A)",
            "K_0(-alpha A, d)",
    };

    for( unsigned u=0; u<outs.size(); u++)
    {
        std::cout << "\n#Compute x = "<<outs[u]<<" b " << std::endl;

        Container x = dg::evaluate(lhs, g), x_exac(x), x_h(x), b(x), error(x);
        Container b_h(b);
        Container one = dg::evaluate(dg::ONE(), g);

        //note that d must fulfill boundary conditions and should be positive definite!
        Container d = dg::evaluate(dg::Cauchy(lx/2., ly/2., 3., 3., amp), g);
        //add constant background field to d
        dg::blas1::plus(d, bgamp);

        std::cout << outs[u] << ":\n";

        dg::mat::UniversalLanczos<Container> krylovfunceigen( x, max_iter);
        dg::mat::UniversalLanczos<Container> krylovfunceigend( x, max_iter);
        dg::mat::ProductMatrixFunction<Container> krylovproduct( x, max_iter);


        dg::mat::GyrolagK<double> func(0, -alpha);
        auto funcE1 = dg::mat::make_FuncEigen_Te1( func);
        double time = 0.;

        //MLanczos-universal
        if (u==0)
        {
            t.tic();
            iter= krylovfunceigen.solve(x, funcE1, A, b, w2d, eps, 1., "universal");
            t.toc();
            time = t.diff();
        }
        if (u==1)
        {
            t.tic();
            //iter = krylovproduct.apply( x, func, d, A, b, w2d, eps, 1.);
            double max = dg::blas1::reduce( d, -1e308, thrust::maximum<double>());
            auto unary_func = dg::mat::make_FuncEigen_Te1( [&](double x) {return func( max, x);});
            auto T = krylovproduct.lanczos().tridiag( unary_func, A, b, w2d, eps, 1.,
                "universal", 1.0, 1);
            iter = T.M.size();
            krylovproduct.compute_vlcl( func, d, A, T, x, b, krylovproduct.lanczos().get_bnorm());
            t.toc();
            time = t.diff();
        }
        if (u==2)
        {
            t.tic();
            //iter = krylovproduct.apply_adjoint( x, func, A, d, b, w2d, eps, 1.);
            double max = dg::blas1::reduce( d, -1e308, thrust::maximum<double>());
            auto unary_func = dg::mat::make_FuncEigen_Te1( [&](double x) {return func( x, max);});
            auto T = krylovproduct.lanczos().tridiag( unary_func, A, b, w2d, eps, 1.,
                "universal", 1.0, 1);
            iter = T.M.size();
            krylovproduct.compute_vlcl_adjoint( func, A, d, T, x, b,
                w2d, krylovproduct.lanczos().get_bnorm());
            t.toc();
            time = t.diff();
        }
        //Compute errors
        if (u==0)
        {
            dg::blas1::scal(x_exac, func(ell_fac));
        }
        else
        {
            Container fd(d); // helper variable
            //Compute absolute and relative error in adjointness
            if (u==2 )
            {
                x_h = dg::evaluate(lhss, g); // -> g
                dg::blas1::axpby(ell_facs, d, 0.0, fd);
                dg::blas1::transform(fd, fd, dg::mat::GyrolagK<double>(0.,-alpha));
                dg::blas1::pointwiseDot(fd, x_h, x_exac); //x_exac = f(-alpha*(ms^2+ns^2) d) sin(x*ms) cos(y*ms) \equiv exp(d,-alpha A) g
                x_h = dg::evaluate(lhs, g); // -> f
                double fOg = dg::blas2::dot( x_h, w2d, x_exac); //<f,exp(d,-alpha A) g>
                std::cout << "#    <f, exp(d,-alpha A) g> = " << fOg << std::endl;
                x_h = dg::evaluate(lhss, g); // -> g
                double gOadjf = dg::blas2::dot( x, w2d, x_h); //<exp(-alpha A, d)f, g>
                std::cout << "#    <exp(-alpha A, d)f, g> = " << gOadjf << std::endl;

                double eabs_adj = fOg-gOadjf; // <f,exp(d,-alpha A) g> -<exp(-alpha A, d)f, g>
                std::cout << "#    Errors in adjointness"<< "\n";
                std::cout << "#    universal-abserror: "<< eabs_adj  << "\n";
                std::cout << "    universal-error: "<< eabs_adj/fOg  << "\n";
            }
            //Compute exact error for product exponential (is used also for adjoint product exponential since we have no analytical solution there)
            x_h = dg::evaluate(lhs, g);
            dg::blas1::axpby(ell_fac, d, 0.0, fd);
            dg::blas1::transform(fd, fd, dg::mat::GyrolagK<double>(0.,-alpha));
            dg::blas1::pointwiseDot(fd, x_h, x_exac); //x_exac = f(-alpha*(m^2+n^2) d) sin(m x) cos(n y)
        }
        std::cout << "    universal-time: "<<time<<"s \n";
        if (u==0 || u==1) {
            dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
            erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));
            std::cout << "    universal-error: "<<erel  << "\n";
        }
        std::cout << "    universal-iter: " <<std::setw(3)<< iter << "\n";
    }

    return 0;
}
